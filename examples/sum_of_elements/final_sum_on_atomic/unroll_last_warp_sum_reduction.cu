#include "hip/hip_runtime.h"
#include <stdio.h>
#include <time.h>
#include <iostream>
#include <chrono>

using namespace std;

#define BLOCK_SIZE 256   // Tune for your GPU (L40S handles 256–512 well)


void fill_array(int N, int max_val, int* x) {
    srand(time(0));
    for (int i = 0; i < N; i++) {
        x[i] = 1;
    }
}

void print_array(int N, int* x) {
    printf("printing first %d elements", N);
    for (int i = 0; i < N; i++) {
        printf("%d ", x[i]);
    }

    printf("\n");
}


void sumCPU(int *in, int *out, int n) {
    for (int i = 0; i < n; i++) {
        (*out) += in[i];
    }
}

__device__ void warpReduceUnrolled(volatile int* s, int tid) {
    s[tid] += s[tid + 32];
    s[tid] += s[tid + 16];
    s[tid] += s[tid + 8];
    s[tid] += s[tid + 4];
    s[tid] += s[tid + 2];
    s[tid] += s[tid + 1];
}

__global__ void unrollLastWarpReduction(int *in, int *out, int n) {
    extern __shared__ int subArray[];
    int tid = threadIdx.x;
    int gid = blockIdx.x * blockDim.x * 2 + threadIdx.x;

    int val1 = (gid < n) ? in[gid] : 0;
    int val2 = (gid + blockDim.x < n) ? in[gid + blockDim.x] : 0;

    subArray[tid] = val1 + val2;
    __syncthreads();

    for (int stride = blockDim.x / 2; stride > 32; stride /= 2) {
        if (tid < stride) {
            subArray[tid] += subArray[tid + stride];
        }
        __syncthreads();
    }

    if (tid < 32) {
        warpReduceUnrolled(subArray, tid);
    }

    if (tid == 0) {
        atomicAdd(out, subArray[0]);
    }
}


int main(int argc, char* argv[]) {
    // Use argument or default to 1024 elements
    int N = (argc > 1) ? atoi(argv[1]) : (1 << 28);

    cout << "Summing " << N << " integers (last-warp unroll)\n";

    // Allocate unified memory
    int *in, *out_gpu;
    hipMallocManaged(&in, N * sizeof(int));
    hipMallocManaged(&out_gpu, sizeof(int));
    *out_gpu = 0;

    fill_array(N, in);


    // --- GPU Reduction ---
    const int THREAD_COUNT = BLOCK_SIZE;
    const int BLOCK_COUNT = (N + THREAD_COUNT * 2 - 1) / (THREAD_COUNT * 2);

    *out_gpu = 0;
    auto start_gpu = chrono::high_resolution_clock::now();
    unrollLastWarpReduction<<<BLOCK_COUNT, THREAD_COUNT, THREAD_COUNT * sizeof(int)>>>(in, out_gpu, N);
    hipDeviceSynchronize();
    auto end_gpu = chrono::high_resolution_clock::now();
    chrono::duration<double> gpu_time = end_gpu - start_gpu;

    cout << "GPU Array sum result: " << *out_gpu << endl;
    cout << "GPU Array sum time  : " << gpu_time.count() << " seconds\n";

    // Cleanup
    hipFree(in);
    hipFree(out_gpu);

    return 0;
}
