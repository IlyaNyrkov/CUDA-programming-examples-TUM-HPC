
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>
#include <iostream>
#include <chrono>

using namespace std;

#define BLOCK_SIZE 256   // Tune for your GPU (L40S handles 256–512 well)


void fill_array(int N, int* x) {
    for (int i = 0; i < N; i++) {
        x[i] = 1;
    }
}

void print_array(int N, int* x) {
    printf("printing first %d elements", N);
    for (int i = 0; i < N; i++) {
        printf("%d ", x[i]);
    }

    printf("\n");
}

void sumCPU(int *input, int *output, int n) {
    for (int i = 0; i < n; i++) {
        (*output) += input[i];
    }
}

template <unsigned int blockSize>
__device__ void warpReduceTemplate(volatile int* s, int tid) {
    if (blockSize >= 64) s[tid] += s[tid + 32];
    if (blockSize >= 32) s[tid] += s[tid + 16];
    if (blockSize >= 16) s[tid] += s[tid + 8];
    if (blockSize >= 8) s[tid] += s[tid + 4];
    if (blockSize >= 4) s[tid] += s[tid + 2];
    if (blockSize >= 2) s[tid] += s[tid + 1];
}

template <unsigned int blockSize>
__global__ void gridStrideReduction(int *in, int *out, int n) {
    extern __shared__ int subArr[];
    int tid = threadIdx.x;
    int gid = blockIdx.x * blockSize * 2 + threadIdx.x;
    int gridSize = blockSize * gridDim.x * 2;
    subArr[tid] = 0;
   
    while (gid < n) { subArr[tid] += in[gid] + in[gid+blockSize]; gid += gridSize; };
    __syncthreads();
    

    // if (blockSize >= 1024) { if (tid < 512) subArr[tid] += subArr[tid + 512]; __syncthreads(); }
    if (blockSize >= 512)  { if (tid < 256) subArr[tid] += subArr[tid + 256]; __syncthreads(); }
    if (blockSize >= 256)  { if (tid < 128) subArr[tid] += subArr[tid + 128]; __syncthreads(); }
    if (blockSize >= 128)  { if (tid < 64)  subArr[tid] += subArr[tid + 64];  __syncthreads(); };

    if (tid < 32) warpReduceTemplate<blockSize>(subArr, tid);
    if (tid == 0) atomicAdd(out, subArr[0]);
}

int main(int argc, char* argv[]) {
    int N = (argc > 1) ? atoi(argv[1]) : (1 << 28);

    cout << "Summing " << N << " integers using CPU and GPU (Grid-stride + Warp unroll)\n";

    // Allocate unified memory
    int *input, *output_gpu;
    int output_cpu = 0;
    hipMallocManaged(&input, N * sizeof(int));
    hipMallocManaged(&output_gpu, sizeof(int));
    *output_gpu = 0;

    fill_array(N, input);

    // --- CPU SUM ---
    auto start_cpu = chrono::high_resolution_clock::now();
    sumCPU(input, &output_cpu, N);
    auto end_cpu = chrono::high_resolution_clock::now();
    chrono::duration<double> cpu_time = end_cpu - start_cpu;

    cout << "CPU Array sum result: " << output_cpu << endl;
    cout << "CPU Array sum time  : " << cpu_time.count() << " seconds" << endl;

    // --- GPU SUM ---
    const int THREAD_COUNT = BLOCK_SIZE;
    const int BLOCK_COUNT = (N + (THREAD_COUNT * 2 - 1)) / (THREAD_COUNT * 2); // 2 elements per thread

    *output_gpu = 0;
    auto start_gpu = chrono::high_resolution_clock::now();
    gridStrideReduction<THREAD_COUNT><<<BLOCK_COUNT, THREAD_COUNT, THREAD_COUNT * sizeof(int)>>>(input, output_gpu, N);
    hipDeviceSynchronize();
    auto end_gpu = chrono::high_resolution_clock::now();
    chrono::duration<double> gpu_time = end_gpu - start_gpu;

    cout << "GPU Array sum result: " << *output_gpu << endl;
    cout << "GPU Array sum time  : " << gpu_time.count() << " seconds" << endl;

    // Cleanup
    hipFree(input);
    hipFree(output_gpu);

    return 0;
}
