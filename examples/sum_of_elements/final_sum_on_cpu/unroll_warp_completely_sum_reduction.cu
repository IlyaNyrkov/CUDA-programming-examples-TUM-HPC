
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>
#include <iostream>
#include <chrono>

using namespace std;

#define BLOCK_SIZE 256  // Tune for your GPU (L40S handles 256–512 well)

void fill_array(int N, int max_val, int* x) {
    srand(time(0));
    for (int i = 0; i < N; i++) {
        x[i] = rand() % (max_val + 1);
    }
}

void print_array(int N, int* x) {
    printf("printing first %d elements: ", N);
    for (int i = 0; i < N; i++) {
        printf("%d ", x[i]);
    }
    printf("\n");
}

void sumCPU(int *input, int *output, int n) {
    for (int i = 0; i < n; i++) {
        (*output) += input[i];
    }
}

template <unsigned int blockSize>
__device__ void warpReduceTemplate(volatile int* s, int tid) {
    if (blockSize >= 64) s[tid] += s[tid + 32];
    if (blockSize >= 32) s[tid] += s[tid + 16];
    if (blockSize >= 16) s[tid] += s[tid + 8];
    if (blockSize >= 8)  s[tid] += s[tid + 4];
    if (blockSize >= 4)  s[tid] += s[tid + 2];
    if (blockSize >= 2)  s[tid] += s[tid + 1];
}

template <unsigned int blockSize>
__global__ void unrollWarpCompletelyReduction(int *input, int *partialSums, int n) {
    extern __shared__ int subArray[];
    int tid = threadIdx.x;
    int gid = blockIdx.x * blockSize * 2 + threadIdx.x;

    int sum = 0;
    if (gid < n) sum += input[gid];
    if (gid + blockSize < n) sum += input[gid + blockSize];

    subArray[tid] = sum;
    __syncthreads();

    if (blockSize >= 128) { if (tid < 64) subArray[tid] += subArray[tid + 64]; __syncthreads(); }

    if (tid < 32) warpReduceTemplate<blockSize>(subArray, tid);

    if (tid == 0) {
        partialSums[blockIdx.x] = subArray[0];
    }
}

int main(int argc, char* argv[]) {
    // Default number of elements or from CLI
    int N = (argc > 1) ? atoi(argv[1]) : (1 << 28);
    const int max_val = 10;

    if (N <= 0) {
        cerr << "Invalid number of elements.\n";
        return 1;
    }

    cout << "Summing " << N << " integers using warp-unroll reduction with CPU final sum...\n";

    // Allocate memory
    int *input;
    hipMallocManaged(&input, N * sizeof(int));
    fill_array(N, max_val, input);
    print_array(min(N, 10), input);

    // ---------------- CPU SUM ----------------
    int output_cpu = 0;
    auto start_cpu = chrono::high_resolution_clock::now();
    sumCPU(input, &output_cpu, N);
    auto end_cpu = chrono::high_resolution_clock::now();
    chrono::duration<double> cpu_time = end_cpu - start_cpu;

    cout << "CPU Array sum result: " << output_cpu << endl;
    cout << "CPU Array sum time  : " << cpu_time.count() << " seconds\n";

    // ---------------- GPU SUM ----------------
    const int THREAD_COUNT = BLOCK_SIZE;
    const int BLOCK_COUNT = (N + THREAD_COUNT * 2 - 1) / (THREAD_COUNT * 2);

    int* partialSums;
    hipMallocManaged(&partialSums, BLOCK_COUNT * sizeof(int));

    auto start_gpu = chrono::high_resolution_clock::now();
    unrollWarpCompletelyReduction<THREAD_COUNT><<<BLOCK_COUNT, THREAD_COUNT, THREAD_COUNT * sizeof(int)>>>(input, partialSums, N);
    hipDeviceSynchronize();

    long long final_sum = 0;
    for (int i = 0; i < BLOCK_COUNT; ++i) {
        final_sum += partialSums[i];
    }
    auto end_gpu = chrono::high_resolution_clock::now();
    chrono::duration<double> gpu_time = end_gpu - start_gpu;

    cout << "GPU Array sum result: " << final_sum << endl;
    cout << "GPU Array sum time  : " << gpu_time.count() << " seconds\n";

    hipFree(input);
    hipFree(partialSums);

    return 0;
}
