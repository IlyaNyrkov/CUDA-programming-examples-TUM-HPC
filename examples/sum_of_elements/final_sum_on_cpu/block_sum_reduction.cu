
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>
#include <iostream>
#include <chrono>

using namespace std;

#define BLOCK_SIZE 256   // Tune for your GPU (L40S handles 256–512 well)

void fill_array(int N, int max_val, int* x) {
    srand(time(0));
    for (int i = 0; i < N; i++) {
        x[i] = rand() % (max_val + 1);
    }
}

void print_array(int N, int* x) {
    printf("printing first %d elements: ", N);
    for (int i = 0; i < N; i++) {
        printf("%d ", x[i]);
    }
    printf("\n");
}

void sumCPU(int *input, int *output, int n) {
    for (int i = 0; i < n; i++) {
        (*output) += input[i];
    }
}

__global__ void blockSumReduction(int *in, int *out, int n) {
    extern __shared__ int subArray[];
    int tid = threadIdx.x;
    int gid = blockIdx.x * blockDim.x * 2 + threadIdx.x;

    subArray[tid] = 0;
    if (gid < n) subArray[tid] += in[gid];
    if (gid + blockDim.x < n) subArray[tid] += in[gid + blockDim.x];
    __syncthreads();

    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (tid < stride) {
            subArray[tid] += subArray[tid + stride];
        }
        __syncthreads();
    }

    if (tid == 0) {
        out[blockIdx.x] = subArray[0]; // Store per-block result
    }
}

int main(int argc, char* argv[]) {
    int N = (argc > 1) ? atoi(argv[1]) : (1 << 28);
    const int max = 10;

    cout << "Summing " << N << " integers using CPU and GPU (Block-wise reduction)\n";

    // Allocate and fill input
    int *input;
    hipMallocManaged(&input, N * sizeof(int));
    fill_array(N, max, input);
    print_array(min(N, 10), input);

    // ------------------ CPU SUM ------------------
    int output_cpu = 0;
    auto start_cpu = chrono::high_resolution_clock::now();
    sumCPU(input, &output_cpu, N);
    auto end_cpu = chrono::high_resolution_clock::now();
    chrono::duration<double> cpu_time = end_cpu - start_cpu;

    cout << "CPU Array sum result: " << output_cpu << endl;
    cout << "CPU Array sum time  : " << cpu_time.count() << " seconds\n";

    // ------------------ GPU SUM ------------------
    const int THREAD_COUNT = BLOCK_SIZE;
    const int BLOCK_COUNT = (N + THREAD_COUNT * 2 - 1) / (THREAD_COUNT * 2);

    int *partial_sums;
    hipMallocManaged(&partial_sums, BLOCK_COUNT * sizeof(int));

    auto start_gpu = chrono::high_resolution_clock::now();
    blockSumReduction<<<BLOCK_COUNT, THREAD_COUNT, THREAD_COUNT * sizeof(int)>>>(input, partial_sums, N);
    hipDeviceSynchronize();

    // Final reduction on CPU
    long long final_sum = 0;
    for (int i = 0; i < BLOCK_COUNT; i++) {
        final_sum += partial_sums[i];
    }
    auto end_gpu = chrono::high_resolution_clock::now();
    chrono::duration<double> gpu_time = end_gpu - start_gpu;

    cout << "GPU Array sum result: " << final_sum << endl;
    cout << "GPU total reduction time: " << gpu_time.count() << " seconds\n";

    // Cleanup
    hipFree(input);
    hipFree(partial_sums);

    return 0;
}
