#include "hip/hip_runtime.h"
#include <stdio.h>
#include <time.h>
#include <iostream>
#include <chrono>

using namespace std;

void fill_array(int N, int max_val, int* x) {
    srand(time(0));
    for (int i = 0; i < N; i++) {
        x[i] = rand() % (max_val + 1);
    }
}

void print_array(int N, int* x) {
    printf("printing first %d elements", N);
    for (int i = 0; i < N; i++) {
        printf("%d ", x[i]);
    }

    printf("\n");
}


void sumCPU(int *input, int *output, int n) {
    for (int i = 0; i < n; i++) {
        (*output) += input[i];
    }
}

__device__ void warpReduceUnrolled(volatile int* s, int tid) {
    s[tid] += s[tid + 32];
    s[tid] += s[tid + 16];
    s[tid] += s[tid + 8];
    s[tid] += s[tid + 4];
    s[tid] += s[tid + 2];
    s[tid] += s[tid + 1];
}

__global__ void unrollLastWarpReduction(int *input, int *output, int n) {
    extern __shared__ int subArray[];
    int tid = threadIdx.x;
    int gid = blockIdx.x * blockDim.x * 2 + threadIdx.x;

    int val1 = 0;
    
    if (gid < n) {
        val1 = array[gid];
    }

    int val2 = 0;

    if (gid + blockDim.x < n) {
        val2 = array[gid + blockDim.x];
    }

    subArray[tid] = val1 + val2;
    __syncthreads();

    for (int stride = blockDim.x / 2; stride > 0; stride /= 2) {
        if (tid < stride) {
            subArray[tid] += subArray[tid + stride];
        }

        __syncthreads();
    }

    if (tid == 0) {
        atomicAdd(output, subArray[0]);
    }
}

int main() {
    const int N = 1 << 10;
    const int max = 100;

    int* input, output_cpu, output_gpu;
    hipMallocManaged(&input, N*sizeof(int));
    fill_array(N, 100, input);
    print_array(10, input);

    auto start_cpu = chrono::high_resolution_clock::now();
    sumCPU(input, output_cpu, N);
    auto end_cpu = chrono::high_resolution_clock::now();

    chrono::duration<double> cpu_time = end_cpu - start_cpu;

    cout << "CPU Array sum time: " << cpu_time.count() << " seconds" << endl;
    cout << "CPU Array sum result: " << *output_cpu << endl; 

    const int THREAD_COUNT = 128;
    const int BLOCK_COUNT = (N + (THREAD_COUNT - 1)) / THREAD_COUNT;

    auto start_gpu = chrono::high_resolution_clock::now();
    unrollLastWarpReduction<<<BLOCK_COUNT, THREAD_COUNT, THREAD_COUNT>>>(input, output_gpu, N);
    hipDeviceSynchronize();
    auto end_gpu = chrono::high_resolution_clock::now();

    chrono::duration<double> gpu_time = end_gpu - start_gpu;

    cout << "GPU Array sum time: " << gpu_time.count() << " seconds" << endl;
    cout << "GPU Array sum result: " << *output_gpu << endl; 


}