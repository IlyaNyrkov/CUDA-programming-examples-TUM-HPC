
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>
#include <chrono>
#include <numeric>


using namespace std;

#define BLOCK_SIZE 256  // L40S/H100 optimal: 256–512

void fill_array(int N, int* x) {
    for (int i = 0; i < N; i++) {
        x[i] = rand() % 100;
    }
}

__device__ void warpReduceUnrolled(volatile int* s, int tid) {
    s[tid] += s[tid + 32];
    s[tid] += s[tid + 16];
    s[tid] += s[tid + 8];
    s[tid] += s[tid + 4];
    s[tid] += s[tid + 2];
    s[tid] += s[tid + 1];
}

__global__ void unrollLastWarpReduction(int *in, int *partialSums) {
    extern __shared__ int subArr[];
    
    unsigned int tid = threadIdx.x;
    unsigned int gid = blockIdx.x * (blockDim.x*2) + threadIdx.x;
    subArr[tid] = in[gid] + in[gid + blockDim.x];
    __syncthreads();

    for (unsigned int stride = blockDim.x / 2; stride > 32; stride >>= 1) {
        if (tid < stride) {
            subArr[tid] += subArr[tid + stride];
        }
        __syncthreads();
    }

    if (tid < 32) warpReduceUnrolled(subArr, tid);

    if (tid == 0) {
        partialSums[blockIdx.x] = subArr[0];
    }
}

int main(int argc, char* argv[]) {
    int N = (argc > 1) ? atoi(argv[1]) : (1 << 26);
    bool verify = (argc > 2 && strcmp(argv[2], "--verify") == 0);

    cout << "Summing " << N << " integers (Last-Warp Unroll + CPU final sum)\n";

    // Allocate host memory
    int *h_input = (int*) malloc(N * sizeof(int));
    fill_array(N, h_input);

    // Allocate device memory
    int *d_input;
    hipMalloc(&d_input, N * sizeof(int));
    hipMemcpy(d_input, h_input, N * sizeof(int), hipMemcpyHostToDevice);

    const int THREAD_COUNT = BLOCK_SIZE;
    const int BLOCK_COUNT = (N + THREAD_COUNT * 2 - 1) / (THREAD_COUNT * 2);

    int *d_partialSums;
    int *h_partialSums = (int*) malloc(BLOCK_COUNT * sizeof(int));
    hipMalloc(&d_partialSums, BLOCK_COUNT * sizeof(int));

    // GPU timing
    auto start_gpu = chrono::high_resolution_clock::now();
    unrollLastWarpReduction<<<BLOCK_COUNT, THREAD_COUNT, THREAD_COUNT * sizeof(int)>>>(d_input, d_partialSums);
    hipDeviceSynchronize();
    auto end_gpu = chrono::high_resolution_clock::now();

    // Copy result back and sum on CPU
    hipMemcpy(h_partialSums, d_partialSums, BLOCK_COUNT * sizeof(int), hipMemcpyDeviceToHost);
    long long gpu_result = 0;
    for (int i = 0; i < BLOCK_COUNT; i++) {
        gpu_result += h_partialSums[i];
    }

    chrono::duration<double> gpu_time = end_gpu - start_gpu;

    // Output
    cout << "GPU Array sum result : " << gpu_result << endl;
    cout << "GPU Array sum time   : " << gpu_time.count() << " seconds\n";

    // Optional verification
    if (verify) {
        long long cpu_result = std::accumulate(h_input, h_input + N, 0LL);
        cout << "CPU Array sum result : " << cpu_result << endl;
        if (cpu_result != gpu_result) {
            cout << "[WARNING] GPU result does not match CPU result!\n";
        } else {
            cout << "[OK] GPU and CPU results match.\n";
        }
    }

    // Cleanup
    hipFree(d_input);
    hipFree(d_partialSums);
    free(h_input);
    free(h_partialSums);

    return 0;
}