
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>
#include <chrono>
#include <cstdlib>
#include <numeric>


using namespace std;

#define BLOCK_SIZE 256  // Optimal for H100: 256–512

void fill_array(int N, int* x) {
    for (int i = 0; i < N; i++) {
        x[i] = rand() % 10;
    }
}

template <unsigned int blockSize>
__device__ void warpReduceTemplate(volatile int* s, int tid) {
    if (blockSize >= 64) s[tid] += s[tid + 32];
    if (blockSize >= 32) s[tid] += s[tid + 16];
    if (blockSize >= 16) s[tid] += s[tid + 8];
    if (blockSize >= 8)  s[tid] += s[tid + 4];
    if (blockSize >= 4)  s[tid] += s[tid + 2];
    if (blockSize >= 2)  s[tid] += s[tid + 1];
}

template <unsigned int blockSize>
__global__ void gridStrideReduction(int *in, int *partialSums, int n) {
    extern __shared__ int subArr[];
    
    unsigned int tid = threadIdx.x;
    unsigned int gid = blockIdx.x * (blockSize*2) + threadIdx.x;
    unsigned int gridSize = blockDim.x * 2 * gridDim.x;
    subArr[tid] = 0;

    while(gid < n) { 
        subArr[tid] += in[gid] + in[gid + blockSize]; 
        gid += gridSize; 
      }
  
    __syncthreads();

    if (blockSize >= 512) { if (tid < 256) subArr[tid] += subArr[tid + 256]; __syncthreads(); }
    if (blockSize >= 256) { if (tid < 128) subArr[tid] += subArr[tid + 128]; __syncthreads(); }
    if (blockSize >= 128) { if (tid < 64) subArr[tid] += subArr[tid + 64]; __syncthreads(); }

    if (tid < 32) warpReduceTemplate<blockSize>(subArr, tid);

    if (tid == 0) {
        partialSums[blockIdx.x] = subArr[0];
    }
}


int main(int argc, char* argv[]) {
    bool verify = false;
    int N = (1 << 26);  

    for (int i = 1; i < argc; i++) {
        if (string(argv[i]) == "--verify") verify = true;
        else N = atoi(argv[i]);
    }

    cout << "Summing " << N << " integers using Grid-stride + Warp unroll (GPU only)" << endl;

    // --- Host memory allocation
    int* h_input = (int*)malloc(N * sizeof(int));
    fill_array(N, h_input);

    // --- Device memory allocation
    int *d_input, *d_partialSums;
    hipMalloc(&d_input, N * sizeof(int));

    const int THREAD_COUNT = BLOCK_SIZE;
    const int BLOCK_COUNT = (N + THREAD_COUNT * 2 - 1) / (THREAD_COUNT * 2);
    hipMalloc(&d_partialSums, BLOCK_COUNT * sizeof(int));

    // Copy input from host to device
    hipMemcpy(d_input, h_input, N * sizeof(int), hipMemcpyHostToDevice);

    // --- Run kernel
    auto start_gpu = chrono::high_resolution_clock::now();
    gridStrideReduction<THREAD_COUNT><<<BLOCK_COUNT, THREAD_COUNT, THREAD_COUNT * sizeof(int)>>>(d_input, d_partialSums, N);
    hipDeviceSynchronize();
    auto end_gpu = chrono::high_resolution_clock::now();

    // --- Copy back results
    int* h_partialSums = (int*)malloc(BLOCK_COUNT * sizeof(int));
    hipMemcpy(h_partialSums, d_partialSums, BLOCK_COUNT * sizeof(int), hipMemcpyDeviceToHost);

    // --- Final reduction on CPU
    long long gpu_result = 0;
    for (int i = 0; i < BLOCK_COUNT; ++i) {
        gpu_result += h_partialSums[i];
    }
    chrono::duration<double> gpu_time = end_gpu - start_gpu;

    cout << "GPU Array sum result       : " << gpu_result << endl;
    cout << "GPU total reduction time   : " << gpu_time.count() << " seconds" << endl;

    if (verify) {
        long long cpu_result = accumulate(h_input, h_input + N, 0LL);
        cout << "CPU Array sum result       : " << cpu_result << endl;
        if (cpu_result != gpu_result) {
            cerr << "ERROR: Mismatch between CPU and GPU results!" << endl;
        } else {
            cout << "✅ Results match." << endl;
        }
    }

    // --- Cleanup
    free(h_input);
    free(h_partialSums);
    hipFree(d_input);
    hipFree(d_partialSums);

    return 0;
}