
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>
#include <chrono>
#include <cstdlib>

using namespace std;

#define BLOCK_SIZE 256  // L40S/H100 optimal: 256–512

void fill_array(int N, int* x) {
    for (int i = 0; i < N; i++) {
        x[i] = rand() % 10;
    }
}

__global__ void firstAddGlobalLoadReduction(int *in, int *partialSums, int n) {
    extern __shared__ int subArr[];
    int tid = threadIdx.x;
    int gid = blockIdx.x * blockDim.x * 2 + threadIdx.x;

    int val1 = (gid < n) ? in[gid] : 0;
    int val2 = (gid + blockDim.x < n) ? in[gid + blockDim.x] : 0;

    subArr[tid] = val1 + val2;
    __syncthreads();

    for (int stride = blockDim.x / 2; stride > 0; stride /= 2) {
        if (tid < stride) {
            subArr[tid] += subArr[tid + stride];
        }
        __syncthreads();
    }

    if (tid == 0) {
        partialSums[blockIdx.x] = subArr[0];
    }
}

int main(int argc, char* argv[]) {
    int N = (argc > 1) ? atoi(argv[1]) : (1 << 28);  // Default: 268M elements

    cout << "Summing " << N << " integers using First-Add-Global-Load (GPU-only + CPU final sum)\n";

    // --- Host allocation
    int *h_input = (int*)malloc(N * sizeof(int));
    fill_array(N, h_input);

    // --- Device allocation
    int *d_input;
    hipMalloc(&d_input, N * sizeof(int));

    const int THREAD_COUNT = BLOCK_SIZE;
    const int BLOCK_COUNT = (N + THREAD_COUNT * 2 - 1) / (THREAD_COUNT * 2);

    int *d_partial_sums;
    hipMalloc(&d_partial_sums, BLOCK_COUNT * sizeof(int));

    // --- Copy input data to device
    hipMemcpy(d_input, h_input, N * sizeof(int), hipMemcpyHostToDevice);

    // --- Launch kernel
    auto start_gpu = chrono::high_resolution_clock::now();
    firstAddGlobalLoadReduction<<<BLOCK_COUNT, THREAD_COUNT, THREAD_COUNT * sizeof(int)>>>(d_input, d_partial_sums, N);
    hipDeviceSynchronize();

    // --- Copy partial results back
    int *h_partial_sums = (int*)malloc(BLOCK_COUNT * sizeof(int));
    hipMemcpy(h_partial_sums, d_partial_sums, BLOCK_COUNT * sizeof(int), hipMemcpyDeviceToHost);

    long long final_sum = 0;
    for (int i = 0; i < BLOCK_COUNT; i++) {
        final_sum += h_partial_sums[i];
    }
    auto end_gpu = chrono::high_resolution_clock::now();
    chrono::duration<double> gpu_time = end_gpu - start_gpu;

    // --- Output results
    cout << "GPU Array sum result       : " << final_sum << endl;
    cout << "GPU total reduction time   : " << gpu_time.count() << " seconds\n";

    // --- Cleanup
    free(h_input);
    free(h_partial_sums);
    hipFree(d_input);
    hipFree(d_partial_sums);

    return 0;
}
