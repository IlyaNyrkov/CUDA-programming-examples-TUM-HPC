
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>
#include <chrono>
#include <cstdlib>

using namespace std;

#define BLOCK_SIZE 256  // L40S/H100 optimal: 256–512

void fill_array(int N, int* x) {
    for (int i = 0; i < N; i++) {
        x[i] = rand() % 10;
    }
}

__global__ void blockSumReduction(int *in, int *out, int n) {
    extern __shared__ int subArray[];
    int tid = threadIdx.x;
    int gid = blockIdx.x * blockDim.x * 2 + threadIdx.x;

    subArray[tid] = 0;
    if (gid < n) subArray[tid] += in[gid];
    if (gid + blockDim.x < n) subArray[tid] += in[gid + blockDim.x];
    __syncthreads();

    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (tid < stride) {
            subArray[tid] += subArray[tid + stride];
        }
        __syncthreads();
    }

    if (tid == 0) {
        out[blockIdx.x] = subArray[0]; // Write per-block sum
    }
}

int main(int argc, char* argv[]) {
    int N = (argc > 1) ? atoi(argv[1]) : (1 << 28);  // Default: 268M ints

    cout << "Summing " << N << " integers using GPU (Block-wise shared memory + final CPU sum)\n";

    // --- Host allocation & initialization
    int *h_input = (int*)malloc(N * sizeof(int));
    fill_array(N, h_input);

    // --- Device allocations
    int *d_input, *d_partial_sums;
    hipMalloc(&d_input, N * sizeof(int));

    const int THREAD_COUNT = BLOCK_SIZE;
    const int BLOCK_COUNT = (N + THREAD_COUNT * 2 - 1) / (THREAD_COUNT * 2);

    hipMalloc(&d_partial_sums, BLOCK_COUNT * sizeof(int));

    // --- Copy data to device
    hipMemcpy(d_input, h_input, N * sizeof(int), hipMemcpyHostToDevice);

    // --- GPU Kernel Launch & Timing
    auto start_gpu = chrono::high_resolution_clock::now();
    blockSumReduction<<<BLOCK_COUNT, THREAD_COUNT, THREAD_COUNT * sizeof(int)>>>(d_input, d_partial_sums, N);
    hipDeviceSynchronize();
    auto end_gpu = chrono::high_resolution_clock::now();

    // --- Copy back partial sums
    int *h_partial_sums = (int*)malloc(BLOCK_COUNT * sizeof(int));
    hipMemcpy(h_partial_sums, d_partial_sums, BLOCK_COUNT * sizeof(int), hipMemcpyDeviceToHost);

    long long final_sum = 0;
    for (int i = 0; i < BLOCK_COUNT; i++) {
        final_sum += h_partial_sums[i];
    }

    chrono::duration<double> gpu_time = end_gpu - start_gpu;

    // --- Output
    cout << "GPU Array sum result       : " << final_sum << endl;
    cout << "Total GPU reduction time   : " << gpu_time.count() << " seconds\n";

    // --- Cleanup
    free(h_input);
    free(h_partial_sums);
    hipFree(d_input);
    hipFree(d_partial_sums);

    return 0;
}
