
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>
#include <chrono>
#include <cstdlib>
#include <numeric>


using namespace std;

#define BLOCK_SIZE 256  // L40S/H100 optimal: 256–512

void fill_array(int N, int* x) {
    for (int i = 0; i < N; i++) {
        x[i] = rand() % 100;
    }
}

__global__ void blockSumReduction(int *in, int *out, int n) {
    extern __shared__ int subArray[];
    
    unsigned int tid = threadIdx.x;
    unsigned int gid = blockIdx.x * blockDim.x + threadIdx.x;
    subArray[tid] = in[gid];
    __syncthreads();

    for (unsigned int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (tid < stride) {
            subArray[tid] += subArray[tid + stride];
        }
        __syncthreads();
    }

    if (tid == 0) {
        out[blockIdx.x] = subArray[0];
    }
}

int main(int argc, char* argv[]) {
    int N = (argc > 1) ? atoi(argv[1]) : (1 << 26);
    bool verify_cpu = (argc > 2 && string(argv[2]) == "--verify");

    cout << "Summing " << N << " integers using GPU (Block-wise shared memory + final CPU sum)\n";

    // --- Host allocation & initialization
    int *h_input = (int*)malloc(N * sizeof(int));
    fill_array(N, h_input);

    // --- Device allocations
    int *d_input, *d_partial_sums;
    hipMalloc(&d_input, N * sizeof(int));

    const int THREAD_COUNT = BLOCK_SIZE;
    const int BLOCK_COUNT = (N + THREAD_COUNT - 1) / (THREAD_COUNT);

    hipMalloc(&d_partial_sums, BLOCK_COUNT * sizeof(int));

    // --- Copy data to device
    hipMemcpy(d_input, h_input, N * sizeof(int), hipMemcpyHostToDevice);

    // --- GPU Kernel Launch & Timing
    auto start_gpu = chrono::high_resolution_clock::now();
    blockSumReduction<<<BLOCK_COUNT, THREAD_COUNT, THREAD_COUNT * sizeof(int)>>>(d_input, d_partial_sums, N);
    hipDeviceSynchronize();
    auto end_gpu = chrono::high_resolution_clock::now();

    // --- Copy back partial sums
    int *h_partial_sums = (int*)malloc(BLOCK_COUNT * sizeof(int));
    hipMemcpy(h_partial_sums, d_partial_sums, BLOCK_COUNT * sizeof(int), hipMemcpyDeviceToHost);

    long long final_sum = 0;
    for (int i = 0; i < BLOCK_COUNT; i++) {
        final_sum += h_partial_sums[i];
    }

    chrono::duration<double> gpu_time = end_gpu - start_gpu;

    // --- Output
    cout << "GPU Array sum result       : " << final_sum << endl;
    cout << "Total GPU reduction time   : " << gpu_time.count() << " seconds\n";

    if (verify_cpu) {
        auto start_cpu = chrono::high_resolution_clock::now();
        long long cpu_sum = accumulate(h_input, h_input + N, 0LL);
        auto end_cpu = chrono::high_resolution_clock::now();
        chrono::duration<double> cpu_time = end_cpu - start_cpu;

        cout << "CPU std::accumulate result : " << cpu_sum << endl;
        cout << "CPU time                   : " << cpu_time.count() << " seconds\n";

        if (cpu_sum != final_sum) {
            cout << "⚠️ WARNING: GPU and CPU results do not match!" << endl;
        } else {
            cout << "✅ GPU result matches CPU!" << endl;
        }
    }

    // --- Cleanup
    free(h_input);
    free(h_partial_sums);
    hipFree(d_input);
    hipFree(d_partial_sums);

    return 0;
}