// naive_matrix_multiplication.cu
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <chrono>

__global__ void tiledMatMul(int* A, int* B, int* C, int N) {
    __shared__ int sharedA[16][16];
    __shared__ int sharedB[16][16];

    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int row = blockIdx.y * 16 + ty;
    int col = blockIdx.x * 16 + tx;

    int sum = 0;
    for (int tile = 0; tile < (N + 15) / 16; tile++) {
        if (row < N && tile * 16 + tx < N)
            sharedA[ty][tx] = A[row * N + tile * 16 + tx];
        else
            sharedA[ty][tx] = 0;

        if (col < N && tile * 16 + ty < N)
            sharedB[ty][tx] = B[(tile * 16 + ty) * N + col];
        else
            sharedB[ty][tx] = 0;

        __syncthreads();

        for (int k = 0; k < 16; k++) {
            sum += sharedA[ty][k] * sharedB[k][tx];
        }

        __syncthreads();
    }

    if (row < N && col < N)
        C[row * N + col] = sum;
}

void cpuMatMul(int *A, int *B, int *C, int N) {
    for (int row = 0; row < N; row++) {
        for (int col = 0; col < N; col++) {
            int sum = 0;
            for (int k = 0; k < N; k++) {
                sum += A[row * N + k] * B[k * N + col];
            }
            C[row * N + col] = sum;
        }
    }
}

bool compareMatricies(int *left, int* right, int N, int M) {
    for (int row = 0; row < N; row++) {
        for (int col = 0; col < N; col++) {
            if (left[row * N + col] != right[row * N + col])  {
                return false;
            }
        }
    }

    return true;
}

int main() {
    const int N = 1024; // Matrix size N x N
    size_t size = N * N * sizeof(int);

    int *matrixACpu = (int *)malloc(size);
    int *matrixBCpu = (int *)malloc(size);
    int *resultMatrixCpu_cpu = (int *)malloc(size);
    int *resultMatrixCpu_gpu = (int *)malloc(size);

    int *matrixAGpu, *matrixBGpu, *matrixCGpu;
    hipMalloc(&matrixAGpu, size);
    hipMalloc(&matrixBGpu, size);
    hipMalloc(&matrixCGpu, size);

    for (int i = 0; i < N * N; i++) {
        matrixACpu[i] = 1;
        matrixBCpu[i] = 1;
    }

    hipMemcpy(matrixAGpu, matrixACpu, size, hipMemcpyHostToDevice);
    hipMemcpy(matrixBGpu, matrixBCpu, size, hipMemcpyHostToDevice);

    dim3 threads(16, 16);
    dim3 blocks((N + threads.x - 1) / threads.x, (N + threads.y - 1) / threads.y);

    auto start_gpu = std::chrono::high_resolution_clock::now();
    tiledMatMul<<<blocks, threads>>>(matrixAGpu, matrixBGpu, matrixCGpu, N);
    hipDeviceSynchronize();
    auto end_gpu = std::chrono::high_resolution_clock::now();

    hipMemcpy(resultMatrixCpu_gpu, matrixCGpu, size, hipMemcpyDeviceToHost);

    auto start_cpu = std::chrono::high_resolution_clock::now();
    cpuMatMul(matrixACpu, matrixBCpu, resultMatrixCpu_cpu, N);
    auto end_cpu = std::chrono::high_resolution_clock::now();

    printf("Matrix %d x %d\n", N, N);
    printf("Tiled Method GPU Time: %f seconds\n", std::chrono::duration<double>(end_gpu - start_gpu).count());
    printf("CPU Time:       %f seconds\n", std::chrono::duration<double>(end_cpu - start_cpu).count());
    printf("Result check:   C[0][0] = %d (GPU), %d (CPU)\n", resultMatrixCpu_gpu[0], resultMatrixCpu_cpu[0]);
    printf("CPU and gpu matricies same: %d\n", compareMatricies(resultMatrixCpu_cpu, resultMatrixCpu_gpu, N, N));

    free(matrixACpu);
    free(matrixBCpu);
    free(resultMatrixCpu_cpu);
    free(resultMatrixCpu_gpu);
    hipFree(matrixAGpu);
    hipFree(matrixBGpu);
    hipFree(matrixCGpu);
    return 0;
}
