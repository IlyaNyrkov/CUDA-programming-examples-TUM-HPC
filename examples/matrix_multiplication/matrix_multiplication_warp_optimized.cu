// naive_matrix_multiplication.cu
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <chrono>

__global__ void warpTiledMatMul(int* A, int* B, int* C, int N) {
    __shared__ int sharedA[32][32];
    __shared__ int sharedB[32][32];

    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int row = blockIdx.y * 32 + ty;
    int col = blockIdx.x * 32 + tx;

    int sum = 0;
    for (int tile = 0; tile < (N + 15) / 32; tile++) {
        if (row < N && tile * 32 + tx < N)
            sharedA[ty][tx] = A[row * N + tile * 32 + tx];
        else
            sharedA[ty][tx] = 0;

        if (col < N && tile * 32 + ty < N)
            sharedB[ty][tx] = B[(tile * 32 + ty) * N + col];
        else
            sharedB[ty][tx] = 0;

        __syncthreads();

        for (int k = 0; k < 32; k++) {
            sum += sharedA[ty][k] * sharedB[k][tx];
        }

        __syncthreads();
    }

    if (row < N && col < N)
        C[row * N + col] = sum;
}

void cpuMatMul(int *A, int *B, int *C, int N) {
    for (int row = 0; row < N; row++) {
        for (int col = 0; col < N; col++) {
            int sum = 0;
            for (int k = 0; k < N; k++) {
                sum += A[row * N + k] * B[k * N + col];
            }
            C[row * N + col] = sum;
        }
    }
}

bool compareMatricies(int *left, int* right, int N, int M) {
    for (int row = 0; row < N; row++) {
        for (int col = 0; col < N; col++) {
            if (left[row * N + col] != right[row * N + col])  {
                return false;
            }
        }
    }

    return true;
}

int main() {
    const int N = 1024; // Matrix size N x N
    size_t size = N * N * sizeof(int);

    int *matrixACpu = (int *)malloc(size);
    int *matrixBCpu = (int *)malloc(size);
    int *resultMatrixCpu_cpu = (int *)malloc(size);
    int *resultMatrixCpu_gpu = (int *)malloc(size);

    int *matrixAGpu, *matrixBGpu, *matrixCGpu;
    hipMalloc(&matrixAGpu, size);
    hipMalloc(&matrixBGpu, size);
    hipMalloc(&matrixCGpu, size);

    for (int i = 0; i < N * N; i++) {
        matrixACpu[i] = 1;
        matrixBCpu[i] = 1;
    }

    hipMemcpy(matrixAGpu, matrixACpu, size, hipMemcpyHostToDevice);
    hipMemcpy(matrixBGpu, matrixBCpu, size, hipMemcpyHostToDevice);

    dim3 threads(32, 32);
    dim3 blocks((N + threads.x - 1) / threads.x, (N + threads.y - 1) / threads.y);

    auto start_gpu = std::chrono::high_resolution_clock::now();
    warpTiledMatMul<<<blocks, threads>>>(matrixAGpu, matrixBGpu, matrixCGpu, N);
    hipDeviceSynchronize();
    auto end_gpu = std::chrono::high_resolution_clock::now();

    hipMemcpy(resultMatrixCpu_gpu, matrixCGpu, size, hipMemcpyDeviceToHost);

    auto start_cpu = std::chrono::high_resolution_clock::now();
    cpuMatMul(matrixACpu, matrixBCpu, resultMatrixCpu_cpu, N);
    auto end_cpu = std::chrono::high_resolution_clock::now();

    printf("Matrix %d x %d\n", N, N);
    printf("Warp optimized Method GPU Time: %f seconds\n", std::chrono::duration<double>(end_gpu - start_gpu).count());
    printf("CPU Time:       %f seconds\n", std::chrono::duration<double>(end_cpu - start_cpu).count());
    printf("Result check:   C[0][0] = %d (GPU), %d (CPU)\n", resultMatrixCpu_gpu[0], resultMatrixCpu_cpu[0]);
    printf("CPU and gpu matricies same: %d\n", compareMatricies(resultMatrixCpu_cpu, resultMatrixCpu_gpu, N, N));

    free(matrixACpu);
    free(matrixBCpu);
    free(resultMatrixCpu_cpu);
    free(resultMatrixCpu_gpu);
    hipFree(matrixAGpu);
    hipFree(matrixBGpu);
    hipFree(matrixCGpu);
    return 0;
}
