#include <stdio.h>
#include <hip/hip_runtime.h>
#include <chrono>

__global__ void naiveMatMul(int *A, int *B, int *C, int N) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < N && col < N) {
        int sum = 0;
        for (int k = 0; k < N; k++) {
            sum += A[row * N + k] * B[k * N + col];
        }
        C[row * N + col] = sum;
    }
}

void fill_matrix(int *mat, int N) {
    for (int i = 0; i < N * N; ++i) {
        mat[i] = rand() % 10;
    }
}

int main(int argc, char* argv[]) {
    int N = (argc > 1) ? atoi(argv[1]) : 512;
    size_t size = N * N * sizeof(int);

    // Allocate pinned host memory for faster transfer
    int *h_A, *h_B, *h_C;
    hipHostMalloc(&h_A, size, hipHostMallocDefault);
    hipHostMalloc(&h_B, size, hipHostMallocDefault);
    hipHostMalloc(&h_C, size, hipHostMallocDefault);

    fill_matrix(h_A, N);
    fill_matrix(h_B, N);

    int *d_A, *d_B, *d_C;
    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);

    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    dim3 threads(16, 16);
    dim3 blocks((N + threads.x - 1) / threads.x,
                (N + threads.y - 1) / threads.y);

    auto start_gpu = std::chrono::high_resolution_clock::now();
    naiveMatMul<<<blocks, threads>>>(d_A, d_B, d_C, N);
    hipDeviceSynchronize();
    auto end_gpu = std::chrono::high_resolution_clock::now();

    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    printf("Naive Matrix Multiplication (%d x %d)\n", N, N);
    printf("GPU Time: %f seconds\n",
           std::chrono::duration<double>(end_gpu - start_gpu).count());
    printf("Sample result: C[0][0] = %d\n", h_C[0]);

    hipHostFree(h_A);
    hipHostFree(h_B);
    hipHostFree(h_C);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}
