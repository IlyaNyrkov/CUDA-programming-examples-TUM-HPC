#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <hipcub/hipcub.hpp> // Include CUB library
#include <chrono>
#include <numeric>
#include <vector>
#include <tuple>
using namespace std;

template <unsigned int blockSize>
__global__ void warpShuffleReduction(float *in, float *partialSums, int n) {
    __shared__ float warpResults[blockSize / 32]; // Shared memory for warp-level results

    unsigned int tid = threadIdx.x;
    unsigned int gid = blockIdx.x * blockDim.x + threadIdx.x;

    float sum = 0.0f;

    // Each thread loads one element
    if (gid < n) {
        sum = in[gid];
    }

    // Perform warp-level reduction using shuffle instructions
    for (int offset = 16; offset > 0; offset /= 2) {
        sum += __shfl_down_sync(0xffffffff, sum, offset);
    }

    // Store the result of each warp in shared memory
    if (tid % 32 == 0) {
        warpResults[tid / 32] = sum;
    }

    __syncthreads();

    // Perform block-level reduction on warp results using a single warp
    if (tid < 32) {
        sum = (tid < blockDim.x / 32) ? warpResults[tid] : 0.0f;
        for (int offset = 16; offset > 0; offset /= 2) {
            sum += __shfl_down_sync(0xffffffff, sum, offset);
        }
        if (tid == 0) {
            partialSums[blockIdx.x] = sum;
        }
    }
}

template <unsigned int blockSize>
__global__ void warpShuffleReductionVectorized(float *in, float *partialSums, int n) {
    __shared__ float warpResults[blockSize / 32]; // Shared memory for warp-level results

    unsigned int tid = threadIdx.x;
    unsigned int gid = blockIdx.x * blockDim.x * 8 + threadIdx.x;

    float sum = 0.0f;

    // Calculate the base index for this thread
    int baseIndex = gid * 4;

    // Load and sum 8 float4 vectors (32 floats)
    for (int i = 0; i < 8; i++) {
        int index = baseIndex + i * blockDim.x * 4; // Step by blockDim.x * 4 for each float4
        if (index < n) {
            float4 data = reinterpret_cast<float4*>(in)[index / 4];
            sum += data.x;
            sum += data.y;
            sum += data.z;
            sum += data.w;
        }
    }

    // Perform warp-level reduction using shuffle instructions
    for (int offset = 16; offset > 0; offset >>= 1) {
        sum += __shfl_down_sync(0xffffffff, sum, offset);
    }

    // Store the result of each warp in shared memory
    if (tid % 32 == 0) {
        warpResults[tid / 32] = sum;
    }

    __syncthreads();

    // Perform block-level reduction on warp results using a single warp
    if (tid < 32) {
        sum = (tid < blockDim.x / 32) ? warpResults[tid] : 0.0f;
        for (int offset = 16; offset > 0; offset >>= 1) {
            sum += __shfl_down_sync(0xffffffff, sum, offset);
        }
        if (tid == 0) {
            partialSums[blockIdx.x] = sum; // Write the final block sum to global memory
        }
    }
}

// Kernel to sum the partial results from all blocks
__global__ void finalReduction(float *partialSums, float *result, int n) {
    float sum = 0.0f;

    // Each thread processes 4 elements at a time using float4
    int tid = threadIdx.x;
    int numThreads = blockDim.x;
    int numVectors = n / 4; // Number of float4 vectors
    int remainder = n % 4; // Remaining elements

    // Process 4 elements at a time using float4
    for (int i = tid; i < numVectors; i += numThreads) {
        float4 data = reinterpret_cast<float4*>(partialSums)[i];
        sum += data.x + data.y + data.z + data.w;
    }

    // Handle remaining elements (if n is not a multiple of 4)
    if (tid < remainder) {
        sum += partialSums[numVectors * 4 + tid];
    }

    // Perform warp-level reduction
    for (int offset = 16; offset > 0; offset /= 2) {
        sum += __shfl_down_sync(0xffffffff, sum, offset);
    }

    // Store the final result
    if (tid == 0) {
        atomicAdd(result, sum);
    }
}

template <typename Kernel>
std::tuple<float, float> benchmarkKernel(Kernel kernel, float *dev_input_data, float *dev_partial_sums, float *dev_result, int n, int blockSize, int factor) {
    // Calculate the number of blocks
    int numBlocks = (n + blockSize * factor - 1) / (blockSize * factor);

    // Create CUDA events for timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Start timing
    hipEventRecord(start);
    kernel<<<numBlocks, blockSize>>>(dev_input_data, dev_partial_sums, n);
    finalReduction<<<1, 32>>>(dev_partial_sums, dev_result, numBlocks);
    hipDeviceSynchronize();
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    // Calculate elapsed time
    float duration = 0.0f;
    hipEventElapsedTime(&duration, start, stop);

    // Retrieve the result from the device
    float result;
    hipMemcpy(&result, dev_result, sizeof(float), hipMemcpyDeviceToHost);

    // Reset device result memory
    hipMemset(dev_result, 0, sizeof(float));

    // Clean up
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return {result, duration};
}

int main() {
    int n = 1 << 25; // 4M elements
    size_t bytes = n * sizeof(float);

    // Host/CPU arrays
    float *host_input_data = new float[n];

    // Device/GPU arrays
    float *dev_input_data, *dev_partial_sums, *dev_result;

    // Initialize data
    srand(42); // Fixed seed
    for (int i = 0; i < n; i++) {
        host_input_data[i] = static_cast<float>(rand()) / RAND_MAX; // Random floats between 0 and 1
    }

    // Allocate memory on GPU
    hipMalloc(&dev_input_data, bytes);
    hipMalloc(&dev_partial_sums, ((n + 511) / 512) * sizeof(float)); // Adjust for 512-thread blocks
    hipMalloc(&dev_result, sizeof(float));
    hipMemset(dev_result, 0, sizeof(float));

    // Copy data to GPU
    hipMemcpy(dev_input_data, host_input_data, bytes, hipMemcpyHostToDevice);

    int blockSize = 512; // Threads per block

    // Map-like structure to store results and durations
    std::vector<std::tuple<std::string, float, float>> results;

    // Perform CPU reduction
    auto cpu_start = std::chrono::high_resolution_clock::now();
    float cpu_result = std::accumulate(host_input_data, host_input_data + n, 0.0f);
    auto cpu_stop = std::chrono::high_resolution_clock::now();
    auto cpu_duration = std::chrono::duration_cast<std::chrono::microseconds>(cpu_stop - cpu_start).count() / 1000.0f;
    results.emplace_back("CPU", cpu_result, cpu_duration);

    // Benchmark different kernels
    {
        auto [result, duration] = benchmarkKernel(warpShuffleReduction<512>, dev_input_data, dev_partial_sums, dev_result, n, blockSize, 1);
        results.emplace_back("Restored", result, duration);
    }
    {
        auto [result, duration] = benchmarkKernel(warpShuffleReductionVectorized<512>, dev_input_data, dev_partial_sums, dev_result, n, blockSize, 8);
        results.emplace_back("Vectorized", result, duration);
    }

    // Display results and verify correctness
    for (const auto &[technique, result, duration] : results) {
        std::cout << "Technique: " << technique << ", Result: " << result
                  << ", Time: " << duration << " ms" << std::endl;
    }

    // Free memory
    hipFree(dev_input_data);
    hipFree(dev_partial_sums);
    hipFree(dev_result);
    delete[] host_input_data;

    return 0;
}