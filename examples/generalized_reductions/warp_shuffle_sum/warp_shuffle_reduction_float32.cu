#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <hipcub/hipcub.hpp> // Include CUB library
#include <chrono>
#include <numeric>
#include <vector>
#include <tuple>
using namespace std;

template <unsigned int blockSize>
__global__ void warpShuffleReduction(float *in, float *partialSums, int n) {
    __shared__ float warpResults[32]; // Shared memory for warp-level results (32 warps per block)

    unsigned int tid = threadIdx.x;
    unsigned int gid = blockIdx.x * blockDim.x + threadIdx.x;

    float sum = 0.0f;

    // Each thread loads one element
    if (gid < n) {
        sum = in[gid];
    }

    // Perform warp-level reduction using shuffle instructions
    for (int offset = 16; offset > 0; offset /= 2) {
        sum += __shfl_down_sync(0xffffffff, sum, offset);
    }

    // Store the result of each warp in shared memory
    if (tid % 32 == 0) {
        warpResults[tid / 32] = sum;
    }

    __syncthreads();

    // Perform block-level reduction on warp results using a single warp
    if (tid < 32) {
        sum = (tid < blockDim.x / 32) ? warpResults[tid] : 0.0f;
        for (int offset = 16; offset > 0; offset /= 2) {
            sum += __shfl_down_sync(0xffffffff, sum, offset);
        }
        if (tid == 0) {
            partialSums[blockIdx.x] = sum;
        }
    }
}

template <unsigned int blockSize>
__global__ void warpShuffleReductionVectorized(float *in, float *partialSums, int n) {
    __shared__ float warpResults[32]; // Shared memory for warp-level results (32 warps per block)

    unsigned int tid = threadIdx.x;
    unsigned int gid = blockIdx.x * blockDim.x * 8 + threadIdx.x;

    float sum = 0.0f;

    // Calculate the base index for this thread
    int baseIndex = gid * 4;

    // Load and sum 8 float4 vectors (32 floats)
    for (int i = 0; i < 8; i++) {
        int index = baseIndex + i * blockDim.x * 4; // Step by blockDim.x * 4 for each float4
        if (index < n) {
            float4 data = reinterpret_cast<float4*>(in)[index / 4];
            sum += data.x;
            sum += data.y;
            sum += data.z;
            sum += data.w;
        }
    }

    // Perform warp-level reduction using shuffle instructions
    for (int offset = 16; offset > 0; offset >>= 1) {
        sum += __shfl_down_sync(0xffffffff, sum, offset);
    }

    // Store the result of each warp in shared memory
    if (tid % 32 == 0) {
        warpResults[tid / 32] = sum;
    }

    __syncthreads();

    // Perform block-level reduction on warp results using a single warp
    if (tid < 32) {
        sum = (tid < blockDim.x / 32) ? warpResults[tid] : 0.0f;
        for (int offset = 16; offset > 0; offset >>= 1) {
            sum += __shfl_down_sync(0xffffffff, sum, offset);
        }
        if (tid == 0) {
            partialSums[blockIdx.x] = sum; // Write the final block sum to global memory
        }
    }
}

// Kernel to sum the partial results from all blocks
__global__ void finalReduction(float *partialSums, float *result, int n) {
    float sum = 0.0f;

    // Each thread processes 4 elements at a time using float4
    int tid = threadIdx.x;
    int numThreads = blockDim.x;
    int numVectors = n / 4; // Number of float4 vectors
    int remainder = n % 4; // Remaining elements

    // Process 4 elements at a time using float4
    for (int i = tid; i < numVectors; i += numThreads) {
        float4 data = reinterpret_cast<float4*>(partialSums)[i];
        sum += data.x + data.y + data.z + data.w;
    }

    // Handle remaining elements (if n is not a multiple of 4)
    if (tid < remainder) {
        sum += partialSums[numVectors * 4 + tid];
    }

    // Perform warp-level reduction
    for (int offset = 16; offset > 0; offset /= 2) {
        sum += __shfl_down_sync(0xffffffff, sum, offset);
    }

    // Store the final result
    if (tid == 0) {
        atomicAdd(result, sum);
    }
}

template <typename Kernel>
std::tuple<float, float> benchmarkKernel(Kernel kernel, float *dev_input_data, float *dev_partial_sums, float *dev_result, int n, int blockSize, int factor) {
    // Calculate the number of blocks
    int numBlocks = (n + blockSize * factor - 1) / (blockSize * factor);

    // Create CUDA events for timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Start timing
    hipEventRecord(start);
    kernel<<<numBlocks, blockSize>>>(dev_input_data, dev_partial_sums, n);
    finalReduction<<<1, 32>>>(dev_partial_sums, dev_result, numBlocks);
    hipDeviceSynchronize();
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    // Calculate elapsed time
    float duration = 0.0f;
    hipEventElapsedTime(&duration, start, stop);

    // Retrieve the result from the device
    float result;
    hipMemcpy(&result, dev_result, sizeof(float), hipMemcpyDeviceToHost);

    // Reset device result memory
    hipMemset(dev_result, 0, sizeof(float));

    // Clean up
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return {result, duration};
}

void displayAndVerifyResults(const std::vector<std::tuple<std::string, float, float>> &results, float cpu_result) {
    bool all_match = true;

    for (const auto &[technique, result, duration] : results) {
        std::cout << "Technique: " << technique << ", Result: " << result
                  << ", Time: " << duration << " ms" << std::endl;

        if (fabs(result - cpu_result) > 1e-5) { // Allow small floating-point error
            all_match = false;
            std::cout << "\033[31mMismatch in " << technique << ": " << result
                      << " (Expected: " << cpu_result << ")\033[0m\n";
        }
    }

    if (all_match) {
        std::cout << "\033[32mVerification successful: All results match.\033[0m\n";
    } else {
        std::cout << "\033[31mVerification failed: Some results do not match.\033[0m\n";
    }
}

int main() {
    int n = 1 << 25; // 4M elements
    size_t bytes = n * sizeof(float);

    // Host/CPU arrays
    float *host_input_data = new float[n];

    // Device/GPU arrays
    float *dev_input_data, *dev_partial_sums, *dev_result;

    // Initialize data
    srand(42); // Fixed seed
    for (int i = 0; i < n; i++) {
        host_input_data[i] = static_cast<float>(rand()) / RAND_MAX; // Random floats between 0 and 1
    }

    // Allocate memory on GPU
    hipMalloc(&dev_input_data, bytes);
    hipMalloc(&dev_partial_sums, ((n + 1023) / 1024) * sizeof(float));
    hipMalloc(&dev_result, sizeof(float));
    hipMemset(dev_result, 0, sizeof(float));

    // Copy data to GPU
    hipMemcpy(dev_input_data, host_input_data, bytes, hipMemcpyHostToDevice);

    int blockSize = 1024; // Threads per block

    // Map-like structure to store results and durations
    std::vector<std::tuple<std::string, float, float>> results;

    // Perform CPU reduction
    auto cpu_start = std::chrono::high_resolution_clock::now();
    float cpu_result = std::accumulate(host_input_data, host_input_data + n, 0.0f);
    auto cpu_stop = std::chrono::high_resolution_clock::now();
    auto cpu_duration = std::chrono::duration_cast<std::chrono::microseconds>(cpu_stop - cpu_start).count() / 1000.0f;
    results.emplace_back("CPU", cpu_result, cpu_duration);

    // Benchmark different kernels
    {
        auto [result, duration] = benchmarkKernel(warpShuffleReduction<1024>, dev_input_data, dev_partial_sums, dev_result, n, blockSize, 1);
        results.emplace_back("Restored", result, duration);
    }
    {
        auto [result, duration] = benchmarkKernel(warpShuffleReductionVectorized<1024>, dev_input_data, dev_partial_sums, dev_result, n, blockSize, 8);
        results.emplace_back("Vectorized", result, duration);
    }

    // CUB reduction
    float *dev_cub_result;
    hipMalloc(&dev_cub_result, sizeof(float));
    hipMemset(dev_cub_result, 0, sizeof(float)); // Initialize to 0

    void *d_temp_storage = nullptr;
    size_t temp_storage_bytes = 0;

    // Determine temporary device storage requirements
    hipcub::DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, dev_input_data, dev_cub_result, n);
    hipMalloc(&d_temp_storage, temp_storage_bytes);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    hipcub::DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, dev_input_data, dev_cub_result, n);
    hipDeviceSynchronize();
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float cub_duration = 0.0f;
    hipEventElapsedTime(&cub_duration, start, stop);
    float cub_result;
    hipMemcpy(&cub_result, dev_cub_result, sizeof(float), hipMemcpyDeviceToHost);
    results.emplace_back("CUB", cub_result, cub_duration);

    // Display results and verify correctness
    displayAndVerifyResults(results, cpu_result);

    // Free memory
    hipFree(dev_input_data);
    hipFree(dev_partial_sums);
    hipFree(dev_result);
    hipFree(dev_cub_result);
    hipFree(d_temp_storage);
    delete[] host_input_data;

    return 0;
}