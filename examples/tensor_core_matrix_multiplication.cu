// cuBLAS handle
hipblasHandle_t handle;
// Creates a cuBLAS handle,
// which is required for 
// all cuBLAS operations
CHECK_CUBLAS(hipblasCreate(&handle));
// Alpha and beta coefficients
float alpha = 1.0f;
float beta = 0.0f;
// Tensor Core GEMM
// hipblasGemmEx performs the 
// generalized matrix multiplication
// C=alpha*(A*B)+beta*C  
// FMA operation with coefficients
/*
HIPBLAS_OP_N: Indicates no transpose 
for matrices A and B.
M, N, K: Dimensions of the matrices.
alpha,beta: Scalars for the computation.
d_A,d_B,d_C: Pointers to matrices.
HIP_R_16F: A/B are in FP16 format.
HIP_R_32F: C is in FP32 format.
CUBLAS_GEMM_DEFAULT_TENSOR_OP: 
Enables Tensor Core acceleration.
*/
CHECK_CUBLAS(hipblasGemmEx(
    handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
    M, N, K,
    &alpha,
    d_A, HIP_R_16F, M,
    d_B, HIP_R_16F, K,
    &beta,
    d_C, HIP_R_32F, M,
    HIP_R_32F,
    CUBLAS_GEMM_DEFAULT_TENSOR_OP));